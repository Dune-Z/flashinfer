#include "hip/hip_runtime.h"
#include "c_wrapper.cuh"
#include <iostream>

extern "C" {
void vec_add(float *lhs, float *rhs, float *res, size_t size) {
    for (size_t i = 0; i < size; ++i) {
        res[i] = lhs[i] + rhs[i];
    }
}

hipError_t singlePrefillWithKVCahceNoSLE(
    __half* q,
    __half* k,
    __half* v,
    __half* o,
    float* tmp,
    unsigned int num_qo_heads,
    unsigned int num_kv_heads,
    unsigned int qo_len,
    unsigned int kv_len,
    unsigned int head_dim,
    bool causal,
    QKVLayout layout = QKVLayout::kNHD,
    RotaryMode rotary_mode = RotaryMode::kNone,
    bool allow_fp16_qk_reduction = false,
    float rope_scale = 1.f, float rope_theta = 1e4,
    hipStream_t stream = nullptr
) {
}
}